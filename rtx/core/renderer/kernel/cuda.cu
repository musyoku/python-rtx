#include "hip/hip_runtime.h"
#include "../../header/enum.h"
#include "../header/bridge.h"
#include "../header/cuda_common.h"
#include "../header/cuda_texture.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include <stdio.h>

hipTextureObject_t* texture_object_pointer;
hipTextureObject_t texture_object_array[30];
hipArray* texture_cuda_array[30];

void rtx_cuda_malloc(void** gpu_array, size_t size)
{
    assert(size > 0);
    hipError_t error = hipMalloc(gpu_array, size);
    // printf("malloc %p\n", *gpu_array);
    hipError_t status = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error at hipMalloc: %s\n", hipGetErrorString(error));
    }
}
void rtx_cuda_malloc_pointer(void**& gpu_array, size_t size)
{
    printf("hipMalloc] %p\n", &gpu_array);
    assert(size > 0);
    hipError_t error = hipMalloc(&gpu_array, size);
    // printf("malloc %p\n", *gpu_array);
    hipError_t status = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error at hipMalloc: %s\n", hipGetErrorString(error));
    }
}
void rtx_cuda_memcpy_host_to_device(void* gpu_array, void* cpu_array, size_t size)
{
    hipError_t error = hipMemcpy(gpu_array, cpu_array, size, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error at hipMemcpyHostToDevice: %s\n", hipGetErrorString(error));
    }
}
void rtx_cuda_memcpy_device_to_host(void* cpu_array, void* gpu_array, size_t size)
{
    hipError_t error = hipMemcpy(cpu_array, gpu_array, size, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error at hipMemcpyDeviceToHost: %s\n", hipGetErrorString(error));
    }
}
void rtx_cuda_free(void** array)
{
    if (*array != NULL) {
        // printf("free %p\n", *array);
        hipError_t error = hipFree(*array);
        if (error != hipSuccess) {
            fprintf(stderr, "CUDA Error at hipFree: %s\n", hipGetErrorString(error));
        }
        *array = NULL;
    }
}
void rtx_cuda_device_reset()
{
    hipDeviceReset();
}
void rtx_cuda_malloc_texture(int unit_index, int width, int height)
{
    hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();

    hipArray*& array = texture_cuda_array[unit_index];
    hipError_t error = hipMallocArray(&array, &desc, width, height);
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error at hipMallocArray: %s\n", hipGetErrorString(error));
    }
    hipMalloc((void**)&texture_object_pointer, sizeof(hipTextureObject_t*) * 30);
}
void rtx_cuda_memcpy_to_texture(int unit_index, int width_offset, int height_offset, void* data, size_t bytes)
{
    hipArray* array = texture_cuda_array[unit_index];
    // hipError_t error = hipMemcpy2D(array, sizeof(float), data, sizeof(float), width_offset, height_offset, hipMemcpyHostToDevice);
    hipError_t error = hipMemcpyToArray(array, 0, 0, data, bytes, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error at hipMemcpyToArray: %s\n", hipGetErrorString(error));
    }
}
void rtx_cuda_bind_texture(int unit_index)
{
    hipArray* array = texture_cuda_array[unit_index];

    hipResourceDesc resource;
    memset(&resource, 0, sizeof(hipResourceDesc));
    resource.resType = hipResourceTypeArray;
    resource.res.array.array = array;

    hipTextureDesc desc;
    memset(&desc, 0, sizeof(hipTextureDesc));
    desc.normalizedCoords = true;
    desc.readMode = hipReadModeElementType;
    desc.filterMode = hipFilterModePoint;
    desc.addressMode[0] = hipAddressModeWrap;
    desc.addressMode[1] = hipAddressModeWrap;
    hipError_t error = hipCreateTextureObject(&texture_object_array[unit_index], &resource, &desc, NULL);
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error at hipCreateTextureObject: %s\n", hipGetErrorString(error));
    }
    printf("%p\n", texture_object_array);
    hipMemcpy(texture_object_pointer, &texture_object_array[unit_index], sizeof(hipTextureObject_t), hipMemcpyHostToDevice);
}
void rtx_cuda_free_texture(int unit_index)
{
    hipArray* array = texture_cuda_array[unit_index];
    hipFreeArray(array);
    array = NULL;
}

size_t rtx_cuda_get_available_shared_memory_bytes()
{
    hipDeviceProp_t dev;
    hipGetDeviceProperties(&dev, 0);
    return dev.sharedMemPerBlock;
}