#include "hip/hip_runtime.h"
#include "../../header/enum.h"
#include "../header/bridge.h"
#include "../header/cuda_common.h"
#include "../header/cuda_texture.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include <stdio.h>

hipTextureObject_t* g_gpu_serialized_mapping_texture_object_array;
hipTextureObject_t g_cpu_serialized_mapping_texture_object_array[RTX_CUDA_MAX_TEXTURE_UNITS];
hipArray* g_gpu_serialized_mapping_texture_cudaArray_ptr_array[RTX_CUDA_MAX_TEXTURE_UNITS];

void rtx_cuda_malloc(void** gpu_array, size_t size)
{
    assert(size > 0);
    cudaCheckError(hipMalloc(gpu_array, size));
}
void rtx_cuda_malloc_pointer(void**& gpu_array, size_t size)
{
    assert(size > 0);
    cudaCheckError(hipMalloc(&gpu_array, size));
}
void rtx_cuda_memcpy_host_to_device(void* gpu_array, void* cpu_array, size_t size)
{
    cudaCheckError(hipMemcpy(gpu_array, cpu_array, size, hipMemcpyHostToDevice));
}
void rtx_cuda_memcpy_device_to_host(void* cpu_array, void* gpu_array, size_t size)
{
    cudaCheckError(hipMemcpy(cpu_array, gpu_array, size, hipMemcpyDeviceToHost));
}
void rtx_cuda_free(void** array_ref)
{
    if (*array_ref != NULL) {
        cudaCheckError(hipFree(*array_ref));
        *array_ref = NULL;
    }
}
void rtx_cuda_device_reset()
{
    hipDeviceReset();
}
void rtx_cuda_malloc_texture(int unit_index, int width, int height)
{
    hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
    hipArray** array = &g_gpu_serialized_mapping_texture_cudaArray_ptr_array[unit_index];
    cudaCheckError(hipMallocArray(array, &desc, width, height));
    cudaCheckError(hipMalloc((void**)&g_gpu_serialized_mapping_texture_object_array, sizeof(hipTextureObject_t) * RTX_CUDA_MAX_TEXTURE_UNITS));
}
void rtx_cuda_memcpy_to_texture(int unit_index, int width_offset, int height_offset, void* data, size_t bytes)
{
    hipArray* array = g_gpu_serialized_mapping_texture_cudaArray_ptr_array[unit_index];
    cudaCheckError(hipMemcpyToArray(array, 0, 0, data, bytes, hipMemcpyHostToDevice));
}
void rtx_cuda_bind_texture(int unit_index)
{
    hipArray* array = g_gpu_serialized_mapping_texture_cudaArray_ptr_array[unit_index];
    hipResourceDesc resource;
    memset(&resource, 0, sizeof(hipResourceDesc));
    resource.resType = hipResourceTypeArray;
    resource.res.array.array = array;

    hipTextureDesc tex;
    memset(&tex, 0, sizeof(hipTextureDesc));
    tex.normalizedCoords = true;
    tex.readMode = hipReadModeElementType;
    tex.filterMode = hipFilterModeLinear;
    tex.addressMode[0] = hipAddressModeWrap;
    tex.addressMode[1] = hipAddressModeWrap;
    cudaCheckError(hipCreateTextureObject(&g_cpu_serialized_mapping_texture_object_array[unit_index], &resource, &tex, NULL));
    cudaCheckError(hipMemcpy(g_gpu_serialized_mapping_texture_object_array, g_cpu_serialized_mapping_texture_object_array, sizeof(hipTextureObject_t) * RTX_CUDA_MAX_TEXTURE_UNITS, hipMemcpyHostToDevice));
}
void rtx_cuda_free_texture(int unit_index)
{
    hipArray* array = g_gpu_serialized_mapping_texture_cudaArray_ptr_array[unit_index];
    cudaCheckError(hipFreeArray(array));
    array = NULL;
}
size_t rtx_cuda_get_available_shared_memory_bytes()
{
    hipDeviceProp_t dev;
    hipGetDeviceProperties(&dev, 0);
    return dev.sharedMemPerBlock;
}
size_t rtx_cuda_get_cudaTextureObject_t_bytes(){
    return sizeof(hipTextureObject_t);
}