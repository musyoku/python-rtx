#include "../../header/enum.h"
#include "../header/hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include <stdio.h>

void rtx_cuda_malloc(void** gpu_array, size_t size)
{
    assert(size > 0);
    hipError_t error = hipMalloc(gpu_array, size);
    // printf("malloc %p\n", *gpu_array);
    hipError_t status = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error at hipMalloc: %s\n", hipGetErrorString(error));
    }
}
void rtx_cuda_memcpy_host_to_device(void* gpu_array, void* cpu_array, size_t size)
{
    hipError_t error = hipMemcpy(gpu_array, cpu_array, size, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error at hipMemcpyHostToDevice: %s\n", hipGetErrorString(error));
    }
}
void rtx_cuda_memcpy_device_to_host(void* cpu_array, void* gpu_array, size_t size)
{
    hipError_t error = hipMemcpy(cpu_array, gpu_array, size, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error at hipMemcpyDeviceToHost: %s\n", hipGetErrorString(error));
    }
}
void rtx_cuda_free(void** array)
{
    if (*array != NULL) {
        // printf("free %p\n", *array);
        hipError_t error = hipFree(*array);
        if (error != hipSuccess) {
            fprintf(stderr, "CUDA Error at hipFree: %s\n", hipGetErrorString(error));
        }
        *array = NULL;
    }
}
void rtx_cuda_device_reset()
{
    hipDeviceReset();
}