#include "../../header/enum.h"
#include "../header/bridge.h"
#include "../header/cuda_linear_memory_texture.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include <stdio.h>

hipTextureObject_t* g_serial_ray_array_texture_object_cpu_ptr = NULL;
hipTextureObject_t* g_serial_face_vertex_index_array_texture_object_cpu_ptr = NULL;
hipTextureObject_t* g_serial_vertex_array_texture_object_cpu_ptr = NULL;
hipTextureObject_t* g_serial_threaded_bvh_array_texture_object_cpu_ptr = NULL;
hipTextureObject_t* g_serial_threaded_bvh_node_array_texture_object_cpu_ptr = NULL;

hipTextureObject_t* g_serial_ray_array_texture_object_gpu_ptr = NULL;
hipTextureObject_t* g_serial_face_vertex_index_array_texture_object_gpu_ptr = NULL;
hipTextureObject_t* g_serial_vertex_array_texture_object_gpu_ptr = NULL;
hipTextureObject_t* g_serial_threaded_bvh_array_texture_object_gpu_ptr = NULL;
hipTextureObject_t* g_serial_threaded_bvh_node_array_texture_object_gpu_ptr = NULL;

void rtx_cuda_malloc_linear_memory_texture_object(
    hipTextureObject_t** texture_object_cpu_ptr_ref,
    hipTextureObject_t** texture_object_gpu_ptr_ref)
{
    *texture_object_cpu_ptr_ref = (hipTextureObject_t*)malloc(sizeof(hipTextureObject_t));
    hipMalloc((void**)texture_object_gpu_ptr_ref, sizeof(hipTextureObject_t));
}
void rtx_cuda_bind_linear_memory_texture_object(
    hipTextureObject_t** texture_object_cpu_ptr_ref,
    hipTextureObject_t** texture_object_gpu_ptr_ref,
    void* buffer,
    size_t bytes, hipChannelFormatKind format)
{
    hipTextureDesc tex;
    memset(&tex, 0, sizeof(hipTextureDesc));

    tex.normalizedCoords = false;
    tex.addressMode[0] = hipAddressModeBorder;
    tex.addressMode[1] = hipAddressModeBorder;
    tex.addressMode[2] = hipAddressModeBorder;
    tex.readMode = hipReadModeElementType;
    tex.filterMode = hipFilterModePoint;

    hipResourceDesc resource;
    memset(&resource, 0, sizeof(hipResourceDesc));
    resource.resType = hipResourceTypeLinear;
    resource.res.linear.devPtr = buffer;
    resource.res.linear.sizeInBytes = bytes;
    resource.res.linear.desc.f = format;
    resource.res.linear.desc.x = 32;
    resource.res.linear.desc.y = 32;
    resource.res.linear.desc.z = 32;
    resource.res.linear.desc.w = 32;
    hipError_t status = hipCreateTextureObject(*texture_object_cpu_ptr_ref, &resource, &tex, NULL);
    if (status != 0) {
        fprintf(stderr, "CUDA Error at hipCreateTextureObject: %s\n", hipGetErrorString(status));
    }
    status = hipMemcpy(*texture_object_gpu_ptr_ref, *texture_object_cpu_ptr_ref, sizeof(hipTextureObject_t), hipMemcpyHostToDevice);
    if (status != 0) {
        fprintf(stderr, "CUDA Error at hipMemcpy: %s\n", hipGetErrorString(status));
    }
}
void rtx_cuda_free_linear_memory_texture_object(
    hipTextureObject_t* texture_object_cpu_ptr_ref,
    hipTextureObject_t* texture_object_gpu_ptr_ref)
{
    if (texture_object_cpu_ptr_ref) {
        free(texture_object_cpu_ptr_ref);
    }
    if (texture_object_gpu_ptr_ref) {
        hipFree(texture_object_gpu_ptr_ref);
    }
}
void rtx_cuda_allocate_linear_memory_texture_objects()
{
    rtx_cuda_malloc_linear_memory_texture_object(&g_serial_ray_array_texture_object_cpu_ptr, &g_serial_ray_array_texture_object_gpu_ptr);
    rtx_cuda_malloc_linear_memory_texture_object(&g_serial_face_vertex_index_array_texture_object_cpu_ptr, &g_serial_face_vertex_index_array_texture_object_gpu_ptr);
    rtx_cuda_malloc_linear_memory_texture_object(&g_serial_vertex_array_texture_object_cpu_ptr, &g_serial_vertex_array_texture_object_gpu_ptr);
    rtx_cuda_malloc_linear_memory_texture_object(&g_serial_threaded_bvh_array_texture_object_cpu_ptr, &g_serial_threaded_bvh_array_texture_object_gpu_ptr);
    rtx_cuda_malloc_linear_memory_texture_object(&g_serial_threaded_bvh_node_array_texture_object_cpu_ptr, &g_serial_threaded_bvh_node_array_texture_object_gpu_ptr);
}
void rtx_cuda_delete_linear_memory_texture_objects()
{
    rtx_cuda_free_linear_memory_texture_object(g_serial_ray_array_texture_object_cpu_ptr, g_serial_ray_array_texture_object_gpu_ptr);
    rtx_cuda_free_linear_memory_texture_object(g_serial_face_vertex_index_array_texture_object_cpu_ptr, g_serial_face_vertex_index_array_texture_object_gpu_ptr);
    rtx_cuda_free_linear_memory_texture_object(g_serial_vertex_array_texture_object_cpu_ptr, g_serial_vertex_array_texture_object_gpu_ptr);
    rtx_cuda_free_linear_memory_texture_object(g_serial_threaded_bvh_array_texture_object_cpu_ptr, g_serial_threaded_bvh_array_texture_object_gpu_ptr);
    rtx_cuda_free_linear_memory_texture_object(g_serial_threaded_bvh_node_array_texture_object_cpu_ptr, g_serial_threaded_bvh_node_array_texture_object_gpu_ptr);
}