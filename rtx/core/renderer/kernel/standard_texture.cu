#include "hip/hip_runtime.h"
#include "../../header/enum.h"
#include "../../header/struct.h"
#include "../header/bridge.h"
#include "../header/cuda_common.h"
#include "../header/cuda_texture.h"
#include "../header/standard_kernel.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include <stdio.h>

__global__ void standard_texture_memory_kernel(
    int ray_array_size,
    int face_vertex_index_array_size,
    int vertex_array_size,
    RTXObject* global_object_array, int object_array_size,
    RTXMaterialAttributeByte* global_material_attribute_byte_array, int material_attribute_byte_array_size,
    RTXThreadedBVH* global_threaded_bvh_array, int threaded_bvh_array_size,
    int threaded_bvh_node_array_size,
    RTXColor* global_color_mapping_array, int color_mapping_array_size,
    hipTextureObject_t* texture_object_array, int texture_object_array_size,
    RTXPixel* global_render_array,
    int num_rays_per_thread,
    int max_bounce,
    int curand_seed)
{
    extern __shared__ unsigned char shared_memory[];
    int thread_id = threadIdx.x;
    hiprandStatePhilox4_32_10_t state;
    hiprand_init(curand_seed, blockIdx.x * blockDim.x + threadIdx.x, 0, &state);

    int offset = 0;
    RTXObject* shared_object_array = (RTXObject*)&shared_memory[offset];
    offset += sizeof(RTXObject) / sizeof(unsigned char) * object_array_size;

    RTXMaterialAttributeByte* shared_material_attribute_byte_array = (RTXMaterialAttributeByte*)&shared_memory[offset];
    offset += sizeof(RTXMaterialAttributeByte) / sizeof(unsigned char) * material_attribute_byte_array_size;

    RTXThreadedBVH* shared_threaded_bvh_array = (RTXThreadedBVH*)&shared_memory[offset];
    offset += sizeof(RTXThreadedBVH) / sizeof(unsigned char) * threaded_bvh_array_size;

    RTXColor* shared_color_mapping_array = (RTXColor*)&shared_memory[offset];
    offset += sizeof(RTXColor) / sizeof(unsigned char) * color_mapping_array_size;

    if (thread_id == 0) {
        for (int k = 0; k < object_array_size; k++) {
            shared_object_array[k] = global_object_array[k];
        }
        for (int k = 0; k < material_attribute_byte_array_size; k++) {
            shared_material_attribute_byte_array[k] = global_material_attribute_byte_array[k];
        }
        for (int k = 0; k < threaded_bvh_array_size; k++) {
            shared_threaded_bvh_array[k] = global_threaded_bvh_array[k];
        }
        for (int k = 0; k < color_mapping_array_size; k++) {
            shared_color_mapping_array[k] = global_color_mapping_array[k];
        }
    }
    __syncthreads();

    const float eps = 0.0000001;
    CUDARay ray;
    // RTXRay ray;
    float3 ray_direction_inv;
    float3 hit_point;
    float3 hit_face_normal;
    float4 hit_va;
    float4 hit_vb;
    float4 hit_vc;
    RTXObject hit_object;

    for (int n = 0; n < num_rays_per_thread; n++) {
        int ray_index = (blockIdx.x * blockDim.x + threadIdx.x) * num_rays_per_thread + n;
        if (ray_index >= ray_array_size) {
            return;
        }

        // ray.direction = tex1Dfetch(ray_texture, ray_index * 2 + 0);
        // ray.origin = tex1Dfetch(ray_texture, ray_index * 2 + 1);

        ray.direction = tex1Dfetch(serial_ray_array_texture_ref, ray_index * 2 + 0);
        ray.origin = tex1Dfetch(serial_ray_array_texture_ref, ray_index * 2 + 1);

        ray_direction_inv.x = 1.0f / ray.direction.x;
        ray_direction_inv.y = 1.0f / ray.direction.y;
        ray_direction_inv.z = 1.0f / ray.direction.z;

        RTXPixel pixel = { 0.0f, 0.0f, 0.0f, 0.0f };
        RTXPixel path_weight = { 1.0f, 1.0f, 1.0f };

        for (int bounce = 0; bounce < max_bounce; bounce++) {
            float min_distance = FLT_MAX;
            bool did_hit_object = false;

            for (int object_index = 0; object_index < object_array_size; object_index++) {
                RTXObject object = shared_object_array[object_index];
                RTXThreadedBVH bvh = shared_threaded_bvh_array[object_index];

                int bvh_current_node_index = 0;
                for (int traversal = 0; traversal < bvh.num_nodes; traversal++) {
                    if (bvh_current_node_index == THREADED_BVH_TERMINAL_NODE) {
                        break;
                    }

                    int index = bvh.node_index_offset + bvh_current_node_index;

                    CUDAThreadedBVHNode node;
                    float4 attributes_float = tex1Dfetch(serial_threaded_bvh_node_array_texture_ref, index * 3 + 0);
                    int4* attributes_integer_ptr = reinterpret_cast<int4*>(&attributes_float);
                    node.hit_node_index = attributes_integer_ptr->x;
                    node.miss_node_index = attributes_integer_ptr->y;
                    node.assigned_face_index_start = attributes_integer_ptr->z;
                    node.assigned_face_index_end = attributes_integer_ptr->w;
                    node.aabb_max = tex1Dfetch(serial_threaded_bvh_node_array_texture_ref, index * 3 + 1);
                    node.aabb_min = tex1Dfetch(serial_threaded_bvh_node_array_texture_ref, index * 3 + 2);

                    bool is_inner_node = node.assigned_face_index_start == -1;
                    if (is_inner_node) {
                        // http://www.cs.utah.edu/~awilliam/box/box.pdf
                        float tmin = ((ray_direction_inv.x < 0 ? node.aabb_max.x : node.aabb_min.x) - ray.origin.x) * ray_direction_inv.x;
                        float tmax = ((ray_direction_inv.x < 0 ? node.aabb_min.x : node.aabb_max.x) - ray.origin.x) * ray_direction_inv.x;
                        float tmp_tmin = ((ray_direction_inv.y < 0 ? node.aabb_max.y : node.aabb_min.y) - ray.origin.y) * ray_direction_inv.y;
                        float tmp_tmax = ((ray_direction_inv.y < 0 ? node.aabb_min.y : node.aabb_max.y) - ray.origin.y) * ray_direction_inv.y;

                        if ((tmin > tmp_tmax) || (tmp_tmin > tmax)) {
                            bvh_current_node_index = node.miss_node_index;
                            continue;
                        }
                        if (tmp_tmin > tmin) {
                            tmin = tmp_tmin;
                        }
                        if (tmp_tmax < tmax) {
                            tmax = tmp_tmax;
                        }
                        tmp_tmin = ((ray_direction_inv.z < 0 ? node.aabb_max.z : node.aabb_min.z) - ray.origin.z) * ray_direction_inv.z;
                        tmp_tmax = ((ray_direction_inv.z < 0 ? node.aabb_min.z : node.aabb_max.z) - ray.origin.z) * ray_direction_inv.z;
                        if ((tmin > tmp_tmax) || (tmp_tmin > tmax)) {
                            bvh_current_node_index = node.miss_node_index;
                            continue;
                        }
                        if (tmp_tmin > tmin) {
                            tmin = tmp_tmin;
                        }
                        if (tmp_tmax < tmax) {
                            tmax = tmp_tmax;
                        }

                        if (tmax < 0.001) {
                            bvh_current_node_index = node.miss_node_index;
                            continue;
                        }
                    } else {
                        int num_assigned_faces = node.assigned_face_index_end - node.assigned_face_index_start + 1;
                        if (object.geometry_type == RTXGeometryTypeStandard) {

                            for (int m = 0; m < num_assigned_faces; m++) {
                                int index = node.assigned_face_index_start + m + object.face_index_offset;

                                const int4 face = tex1Dfetch(serial_face_vertex_index_array_texture_ref, index);

                                const float4 va = tex1Dfetch(serial_vertex_array_texture_ref, face.x + object.vertex_index_offset);
                                const float4 vb = tex1Dfetch(serial_vertex_array_texture_ref, face.y + object.vertex_index_offset);
                                const float4 vc = tex1Dfetch(serial_vertex_array_texture_ref, face.z + object.vertex_index_offset);

                                float3 edge_ba;
                                edge_ba.x = vb.x - va.x;
                                edge_ba.y = vb.y - va.y;
                                edge_ba.z = vb.z - va.z;

                                float3 edge_ca;
                                edge_ca.x = vc.x - va.x;
                                edge_ca.y = vc.y - va.y;
                                edge_ca.z = vc.z - va.z;

                                float3 h;
                                h.x = ray.direction.y * edge_ca.z - ray.direction.z * edge_ca.y;
                                h.y = ray.direction.z * edge_ca.x - ray.direction.x * edge_ca.z;
                                h.z = ray.direction.x * edge_ca.y - ray.direction.y * edge_ca.x;
                                float f = edge_ba.x * h.x + edge_ba.y * h.y + edge_ba.z * h.z;
                                if (f > -eps && f < eps) {
                                    continue;
                                }

                                f = 1.0f / f;

                                float3 s;
                                s.x = ray.origin.x - va.x;
                                s.y = ray.origin.y - va.y;
                                s.z = ray.origin.z - va.z;
                                float dot = s.x * h.x + s.y * h.y + s.z * h.z;
                                float u = f * dot;
                                if (u < 0.0f || u > 1.0f) {
                                    continue;
                                }

                                h.x = s.y * edge_ba.z - s.z * edge_ba.y;
                                h.y = s.z * edge_ba.x - s.x * edge_ba.z;
                                h.z = s.x * edge_ba.y - s.y * edge_ba.x;
                                dot = h.x * ray.direction.x + h.y * ray.direction.y + h.z * ray.direction.z;
                                float v = f * dot;
                                if (v < 0.0f || u + v > 1.0f) {
                                    continue;
                                }
                                s.x = edge_ba.y * edge_ca.z - edge_ba.z * edge_ca.y;
                                s.y = edge_ba.z * edge_ca.x - edge_ba.x * edge_ca.z;
                                s.z = edge_ba.x * edge_ca.y - edge_ba.y * edge_ca.x;

                                float norm = sqrtf(s.x * s.x + s.y * s.y + s.z * s.z) + 1e-12;

                                s.x = s.x / norm;
                                s.y = s.y / norm;
                                s.z = s.z / norm;

                                dot = s.x * ray.direction.x + s.y * ray.direction.y + s.z * ray.direction.z;
                                if (dot > 0.0f) {
                                    continue;
                                }

                                dot = edge_ca.x * h.x + edge_ca.y * h.y + edge_ca.z * h.z;
                                float t = f * dot;

                                if (t <= 0.001f) {
                                    continue;
                                }
                                if (min_distance <= t) {
                                    continue;
                                }

                                min_distance = t;
                                hit_point.x = ray.origin.x + t * ray.direction.x;
                                hit_point.y = ray.origin.y + t * ray.direction.y;
                                hit_point.z = ray.origin.z + t * ray.direction.z;

                                hit_face_normal.x = s.x;
                                hit_face_normal.y = s.y;
                                hit_face_normal.z = s.z;

                                hit_va = va;
                                hit_vb = vb;
                                hit_vc = vc;

                                did_hit_object = true;
                                hit_object = object;
                            }
                        } else if (object.geometry_type == RTXGeometryTypeSphere) {
                            int index = node.assigned_face_index_start + object.face_index_offset;

                            const int4 face = tex1Dfetch(serial_face_vertex_index_array_texture_ref, index);

                            const float4 center = tex1Dfetch(serial_vertex_array_texture_ref, face.x + object.vertex_index_offset);
                            const float4 radius = tex1Dfetch(serial_vertex_array_texture_ref, face.y + object.vertex_index_offset);

                            float4 oc;
                            oc.x = ray.origin.x - center.x;
                            oc.y = ray.origin.y - center.y;
                            oc.z = ray.origin.z - center.z;

                            const float a = ray.direction.x * ray.direction.x + ray.direction.y * ray.direction.y + ray.direction.z * ray.direction.z;
                            const float b = 2.0f * (ray.direction.x * oc.x + ray.direction.y * oc.y + ray.direction.z * oc.z);
                            const float c = (oc.x * oc.x + oc.y * oc.y + oc.z * oc.z) - radius.x * radius.x;
                            const float d = b * b - 4.0f * a * c;

                            if (d <= 0) {
                                continue;
                            }
                            const float root = sqrt(d);
                            float t = (-b - root) / (2.0f * a);
                            if (t <= 0.001f) {
                                t = (-b + root) / (2.0f * a);
                                if (t <= 0.001f) {
                                    continue;
                                }
                            }

                            if (min_distance <= t) {
                                continue;
                            }
                            min_distance = t;
                            hit_point.x = ray.origin.x + t * ray.direction.x;
                            hit_point.y = ray.origin.y + t * ray.direction.y;
                            hit_point.z = ray.origin.z + t * ray.direction.z;

                            float4 tmp;
                            tmp.x = hit_point.x - center.x;
                            tmp.y = hit_point.y - center.y;
                            tmp.z = hit_point.z - center.z;
                            const float norm = sqrtf(tmp.x * tmp.x + tmp.y * tmp.y + tmp.z * tmp.z) + 1e-12;

                            hit_face_normal.x = tmp.x / norm;
                            hit_face_normal.y = tmp.y / norm;
                            hit_face_normal.z = tmp.z / norm;

                            did_hit_object = true;
                            hit_object = object;
                        }
                    }

                    if (node.hit_node_index == THREADED_BVH_TERMINAL_NODE) {
                        bvh_current_node_index = node.miss_node_index;
                    } else {
                        bvh_current_node_index = node.hit_node_index;
                    }
                }
            }

            RTXPixel hit_color;
            bool did_hit_light = false;
            if (did_hit_object) {
                int material_type = hit_object.layerd_material_types.outside;
                int mapping_type = hit_object.mapping_type;
                int geometry_type = hit_object.geometry_type;

                if (mapping_type == RTXMappingTypeSolidColor) {
                    RTXColor color = shared_color_mapping_array[hit_object.mapping_index];
                    hit_color.r = color.r;
                    hit_color.g = color.g;
                    hit_color.b = color.b;
                } else if (mapping_type == RTXMappingTypeTexture) {
                    if (geometry_type == RTXGeometryTypeStandard) {
                        // compute barycentric coordinates
                        // https://shikihuiku.wordpress.com/2017/05/23/barycentric-coordinates%E3%81%AE%E8%A8%88%E7%AE%97%E3%81%A8perspective-correction-partial-derivative%E3%81%AB%E3%81%A4%E3%81%84%E3%81%A6/
                        float3 d1 = {
                            hit_va.x - hit_vc.x,
                            hit_va.y - hit_vc.y,
                            hit_va.z - hit_vc.z
                        };
                        float3 d2 = {
                            hit_vb.x - hit_vc.x,
                            hit_vb.y - hit_vc.y,
                            hit_vb.z - hit_vc.z
                        };
                        float3 d = {
                            hit_point.x - hit_vc.x,
                            hit_point.y - hit_vc.y,
                            hit_point.z - hit_vc.z
                        };
                        float d1x = d1.x * d1.x + d1.y * d1.y + d1.z * d1.z;
                        float d1y = d1.x * d2.x + d1.y * d2.y + d1.z * d2.z;
                        float d2x = d1y;
                        float d2y = d2.x * d2.x + d2.y * d2.y + d2.z * d2.z;
                        float dx = d.x * d1.x + d.y * d1.y + d.z * d1.z;
                        float dy = d.x * d2.x + d.y * d2.y + d.z * d2.z;

                        float det = d1x * d2y - d1y * d2x;
                        float3 lambda = {
                            (dx * d2y - dy * d2x) / det,
                            (d1x * dy - d1y * dx) / det,
                            0.0f
                        };
                        lambda.z = 1.0f - lambda.x - lambda.y;

                        float div_w = 1.0f / hit_va.w * lambda.x + 1.0f / hit_vb.w * lambda.y + 1.0f / hit_vc.w * lambda.z;

                        float x = lambda.z / div_w;
                        float y = lambda.y / div_w;

                        float4 color = tex2D<float4>(texture_object_array[hit_object.mapping_index], x, y);

                        hit_color.r = color.x;
                        hit_color.g = color.y;
                        hit_color.b = color.z;
                    } else {
                        hit_color.r = 0.0f;
                        hit_color.g = 0.0f;
                        hit_color.b = 0.0f;
                    }
                }

                if (material_type == RTXMaterialTypeLambert) {
                    RTXLambertMaterialAttribute attr = ((RTXLambertMaterialAttribute*)&shared_material_attribute_byte_array[hit_object.material_attribute_byte_array_offset])[0];
                    hit_color.r *= attr.albedo;
                    hit_color.g *= attr.albedo;
                    hit_color.b *= attr.albedo;
                } else if (material_type == RTXMaterialTypeEmissive) {
                    RTXEmissiveMaterialAttribute attr = ((RTXEmissiveMaterialAttribute*)&shared_material_attribute_byte_array[hit_object.material_attribute_byte_array_offset])[0];
                    did_hit_light = true;
                    hit_color.r *= attr.brightness;
                    hit_color.g *= attr.brightness;
                    hit_color.b *= attr.brightness;
                }
            }

            if (did_hit_light) {
                pixel.r += hit_color.r * path_weight.r;
                pixel.g += hit_color.g * path_weight.g;
                pixel.b += hit_color.b * path_weight.b;
            }

            if (did_hit_object) {
                float4 path;
                path.x = hit_point.x - ray.origin.x;
                path.y = hit_point.y - ray.origin.y;
                path.z = hit_point.z - ray.origin.z;
                float distance = sqrt(path.x * path.x + path.y * path.y + path.z * path.z);

                ray.origin.x = hit_point.x;
                ray.origin.y = hit_point.y;
                ray.origin.z = hit_point.z;

                // diffuse reflection
                float unit_diffuese_x = hiprand_normal(&state);
                float unit_diffuese_y = hiprand_normal(&state);
                float unit_diffuese_z = hiprand_normal(&state);
                float norm = sqrt(unit_diffuese_x * unit_diffuese_x + unit_diffuese_y * unit_diffuese_y + unit_diffuese_z * unit_diffuese_z);
                unit_diffuese_x /= norm;
                unit_diffuese_y /= norm;
                unit_diffuese_z /= norm;

                float cosine_term = hit_face_normal.x * unit_diffuese_x + hit_face_normal.y * unit_diffuese_y + hit_face_normal.z * unit_diffuese_z;
                if (cosine_term < 0.0f) {
                    unit_diffuese_x *= -1;
                    unit_diffuese_y *= -1;
                    unit_diffuese_z *= -1;
                    cosine_term *= -1;
                }
                ray.direction.x = unit_diffuese_x;
                ray.direction.y = unit_diffuese_y;
                ray.direction.z = unit_diffuese_z;

                ray_direction_inv.x = 1.0f / ray.direction.x;
                ray_direction_inv.y = 1.0f / ray.direction.y;
                ray_direction_inv.z = 1.0f / ray.direction.z;

                path_weight.r *= 4.0 * hit_color.r * cosine_term;
                path_weight.g *= 4.0 * hit_color.g * cosine_term;
                path_weight.b *= 4.0 * hit_color.b * cosine_term;
            }
        }

        global_render_array[ray_index] = pixel;
    }
}

void rtx_cuda_launch_standard_texture_memory_kernel(
    RTXRay*& gpu_ray_array, int ray_array_size,
    RTXFace*& gpu_face_vertex_index_array, int face_vertex_index_array_size,
    RTXVertex*& gpu_vertex_array, int vertex_array_size,
    RTXObject*& gpu_object_array, int object_array_size,
    RTXMaterialAttributeByte*& gpu_material_attribute_byte_array, int material_attribute_byte_array_size,
    RTXThreadedBVH*& gpu_threaded_bvh_array, int threaded_bvh_array_size,
    RTXThreadedBVHNode*& gpu_threaded_bvh_node_array, int threaded_bvh_node_array_size,
    RTXColor*& gpu_color_mapping_array, int color_mapping_array_size,
    RTXPixel*& gpu_render_array, int render_array_size,
    int num_threads,
    int num_blocks,
    int num_rays_per_thread,
    size_t shared_memory_bytes,
    int max_bounce,
    int curand_seed)
{
    assert(gpu_ray_array != NULL);
    assert(gpu_face_vertex_index_array != NULL);
    assert(gpu_vertex_array != NULL);
    assert(gpu_object_array != NULL);
    assert(gpu_material_attribute_byte_array != NULL);
    assert(gpu_threaded_bvh_array != NULL);
    assert(gpu_threaded_bvh_node_array != NULL);
    assert(gpu_render_array != NULL);
    if (color_mapping_array_size > 0) {
        assert(gpu_color_mapping_array != NULL);
    }

    hipBindTexture(0, serial_ray_array_texture_ref, gpu_ray_array, hipCreateChannelDesc<float4>(), sizeof(RTXRay) * ray_array_size);
    hipBindTexture(0, serial_face_vertex_index_array_texture_ref, gpu_face_vertex_index_array, hipCreateChannelDesc<int4>(), sizeof(RTXFace) * face_vertex_index_array_size);
    hipBindTexture(0, serial_vertex_array_texture_ref, gpu_vertex_array, hipCreateChannelDesc<float4>(), sizeof(RTXVertex) * vertex_array_size);
    hipBindTexture(0, serial_threaded_bvh_node_array_texture_ref, gpu_threaded_bvh_node_array, hipCreateChannelDesc<float4>(), sizeof(RTXThreadedBVHNode) * threaded_bvh_node_array_size);

    standard_texture_memory_kernel<<<num_blocks, num_threads, shared_memory_bytes>>>(
        ray_array_size,
        face_vertex_index_array_size,
        vertex_array_size,
        gpu_object_array, object_array_size,
        gpu_material_attribute_byte_array, material_attribute_byte_array_size,
        gpu_threaded_bvh_array, threaded_bvh_array_size,
        threaded_bvh_node_array_size,
        gpu_color_mapping_array, color_mapping_array_size,
        texture_object_pointer, 30,
        gpu_render_array,
        num_rays_per_thread,
        max_bounce,
        curand_seed);

    cudaCheckError(hipGetLastError());
    cudaCheckError(hipDeviceSynchronize());

    hipUnbindTexture(serial_ray_array_texture_ref);
    hipUnbindTexture(serial_face_vertex_index_array_texture_ref);
    hipUnbindTexture(serial_vertex_array_texture_ref);
    hipUnbindTexture(serial_threaded_bvh_node_array_texture_ref);
}