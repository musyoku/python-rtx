#include "hip/hip_runtime.h"
#include "../../../class/enum.h"
#include "../header/ray_tracing.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include <stdio.h>
#include <time.h>

__global__ void test_kernel(const float* vertices, int num_vertices)
{
    float sum = 0;
    for (int n = 0; n < num_vertices; n++) {
        float x = vertices[n * 4 + 0];
        float y = vertices[n * 4 + 1];
        float z = vertices[n * 4 + 2];
        sum += x + y + z;
    }
}

__global__ void render(
    const float* rays,
    const float* face_vertices,
    const float* face_colors,
    const int* object_types,
    const int* material_types,
    float* color_per_ray,
    const float* camera_inv_matrix,
    const int num_rays_per_thread,
    const int thread_offset,
    const int num_rays,
    const int num_faces,
    const int faces_stride,
    const int colors_stride,
    const int max_path_depth)
{
    unsigned int tid = threadIdx.x;
    hiprandStateXORWOW_t state;
    hiprand_init(0, blockIdx.x * blockDim.x + threadIdx.x, 0, &state);

    __shared__ float shared_face_vertices[41 * 12];
    __shared__ float shared_face_colors[41 * 3];
    __shared__ int shared_object_types[41];
    __shared__ int shared_material_types[41];
    __shared__ float shared_camera_inv_matrix[4][4];

    if (threadIdx.x == 0) {
        for (int n = 0; n < num_faces; n++) {
            for (int s = 0; s < faces_stride; s++) {
                shared_face_vertices[n * faces_stride + s] = face_vertices[n * faces_stride + s];
            }
            for (int s = 0; s < colors_stride; s++) {
                shared_face_colors[n * colors_stride + s] = face_colors[n * colors_stride + s];
            }
            shared_object_types[n] = object_types[n];
            shared_material_types[n] = material_types[n];
        }
        shared_camera_inv_matrix[0][0] = camera_inv_matrix[0];
        shared_camera_inv_matrix[0][1] = camera_inv_matrix[1];
        shared_camera_inv_matrix[0][2] = camera_inv_matrix[2];
        shared_camera_inv_matrix[0][3] = camera_inv_matrix[3];
        shared_camera_inv_matrix[1][0] = camera_inv_matrix[4];
        shared_camera_inv_matrix[1][1] = camera_inv_matrix[5];
        shared_camera_inv_matrix[1][2] = camera_inv_matrix[6];
        shared_camera_inv_matrix[1][3] = camera_inv_matrix[7];
        shared_camera_inv_matrix[2][0] = camera_inv_matrix[8];
        shared_camera_inv_matrix[2][1] = camera_inv_matrix[9];
        shared_camera_inv_matrix[2][2] = camera_inv_matrix[10];
        shared_camera_inv_matrix[2][3] = camera_inv_matrix[11];
        shared_camera_inv_matrix[3][0] = camera_inv_matrix[12];
        shared_camera_inv_matrix[3][1] = camera_inv_matrix[13];
        shared_camera_inv_matrix[3][2] = camera_inv_matrix[14];
        shared_camera_inv_matrix[3][3] = camera_inv_matrix[15];
    }
    __syncthreads();

    for (int n = 0; n < num_rays_per_thread; n++) {
        unsigned int ray_index = (blockIdx.x * blockDim.x + threadIdx.x) * num_rays_per_thread + n + thread_offset;
        if (ray_index >= num_rays) {
            return;
        }

        const int p = ray_index * 7;
        float ray_direction_x = rays[p + 0];
        float ray_direction_y = rays[p + 1];
        float ray_direction_z = rays[p + 2];
        float ray_origin_x = rays[p + 3];
        float ray_origin_y = rays[p + 4];
        float ray_origin_z = rays[p + 5];
        float ray_direction_inv_x = 1.0f / ray_direction_x;
        float ray_direction_inv_y = 1.0f / ray_direction_y;
        float ray_direction_inv_z = 1.0f / ray_direction_z;

        float color_r = 0.0;
        float color_g = 0.0;
        float color_b = 0.0;

        int object_type = 0;
        int material_type = 0;
        float hit_point_x = 0.0f;
        float hit_point_y = 0.0f;
        float hit_point_z = 0.0f;
        float hit_color_r = 0.0f;
        float hit_color_g = 0.0f;
        float hit_color_b = 0.0f;
        float hit_face_normal_x = 0.0f;
        float hit_face_normal_y = 0.0f;
        float hit_face_normal_z = 0.0f;

        color_r = 1.0f;
        color_g = 1.0f;
        color_b = 1.0f;

        const float eps = 0.0000001;
        float reflection_decay_r = 1.0f;
        float reflection_decay_g = 1.0f;
        float reflection_decay_b = 1.0f;
        bool did_hit_light = false;

        for (int depth = 0; depth < max_path_depth; depth++) {
            float min_distance = FLT_MAX;
            bool did_hit_object = false;

            for (int face_index = 0; face_index < num_faces; face_index++) {
                object_type = shared_object_types[face_index];
                const int index = face_index * faces_stride;

                if (object_type == RTX_GEOMETRY_TYPE_STANDARD) {
                    const float va_x = shared_face_vertices[index + 0];
                    const float va_y = shared_face_vertices[index + 1];
                    const float va_z = shared_face_vertices[index + 2];

                    const float vb_x = shared_face_vertices[index + 4];
                    const float vb_y = shared_face_vertices[index + 5];
                    const float vb_z = shared_face_vertices[index + 6];

                    const float vc_x = shared_face_vertices[index + 8];
                    const float vc_y = shared_face_vertices[index + 9];
                    const float vc_z = shared_face_vertices[index + 10];

                    const float edge_ba_x = vb_x - va_x;
                    const float edge_ba_y = vb_y - va_y;
                    const float edge_ba_z = vb_z - va_z;

                    const float edge_ca_x = vc_x - va_x;
                    const float edge_ca_y = vc_y - va_y;
                    const float edge_ca_z = vc_z - va_z;

                    const float h_x = ray_direction_y * edge_ca_z - ray_direction_z * edge_ca_y;
                    const float h_y = ray_direction_z * edge_ca_x - ray_direction_x * edge_ca_z;
                    const float h_z = ray_direction_x * edge_ca_y - ray_direction_y * edge_ca_x;
                    const float a = edge_ba_x * h_x + edge_ba_y * h_y + edge_ba_z * h_z;
                    if (a > -eps && a < eps) {
                        continue;
                    }
                    const float f = 1.0f / a;

                    const float s_x = ray_origin_x - va_x;
                    const float s_y = ray_origin_y - va_y;
                    const float s_z = ray_origin_z - va_z;
                    float dot = s_x * h_x + s_y * h_y + s_z * h_z;
                    const float u = f * dot;
                    if (u < 0.0f || u > 1.0f) {
                        continue;
                    }
                    const float q_x = s_y * edge_ba_z - s_z * edge_ba_y;
                    const float q_y = s_z * edge_ba_x - s_x * edge_ba_z;
                    const float q_z = s_x * edge_ba_y - s_y * edge_ba_x;
                    dot = q_x * ray_direction_x + q_y * ray_direction_y + q_z * ray_direction_z;
                    const float v = f * dot;
                    if (v < 0.0f || u + v > 1.0f) {
                        continue;
                    }
                    float tmp_x = edge_ba_y * edge_ca_z - edge_ba_z * edge_ca_y;
                    float tmp_y = edge_ba_z * edge_ca_x - edge_ba_x * edge_ca_z;
                    float tmp_z = edge_ba_x * edge_ca_y - edge_ba_y * edge_ca_x;

                    float norm = sqrtf(tmp_x * tmp_x + tmp_y * tmp_y + tmp_z * tmp_z) + 1e-12;

                    tmp_x = tmp_x / norm;
                    tmp_y = tmp_y / norm;
                    tmp_z = tmp_z / norm;

                    dot = tmp_x * ray_direction_x + tmp_y * ray_direction_y + tmp_z * ray_direction_z;
                    if (dot > 0.0f) {
                        continue;
                    }

                    dot = edge_ca_x * q_x + edge_ca_y * q_y + edge_ca_z * q_z;
                    const float t = f * dot;

                    if (t <= 0.001f) {
                        continue;
                    }
                    if (min_distance <= t) {
                        continue;
                    }

                    min_distance = t;
                    hit_point_x = ray_origin_x + t * ray_direction_x;
                    hit_point_y = ray_origin_y + t * ray_direction_y;
                    hit_point_z = ray_origin_z + t * ray_direction_z;

                    hit_face_normal_x = tmp_x;
                    hit_face_normal_y = tmp_y;
                    hit_face_normal_z = tmp_z;

                    material_type = shared_material_types[face_index];

                    hit_color_r = shared_face_colors[face_index * colors_stride + 0];
                    hit_color_g = shared_face_colors[face_index * colors_stride + 1];
                    hit_color_b = shared_face_colors[face_index * colors_stride + 2];

                    did_hit_object = true;
                    continue;
                }
                if (object_type == RTX_GEOMETRY_TYPE_SPHERE) {
                    const float center_x = shared_face_vertices[index + 0];
                    const float center_y = shared_face_vertices[index + 1];
                    const float center_z = shared_face_vertices[index + 2];
                    const float radius = shared_face_vertices[index + 4];

                    const float oc_x = ray_origin_x - center_x;
                    const float oc_y = ray_origin_y - center_y;
                    const float oc_z = ray_origin_z - center_z;

                    const float a = ray_direction_x * ray_direction_x + ray_direction_y * ray_direction_y + ray_direction_z * ray_direction_z;
                    const float b = 2.0f * (ray_direction_x * oc_x + ray_direction_y * oc_y + ray_direction_z * oc_z);
                    const float c = (oc_x * oc_x + oc_y * oc_y + oc_z * oc_z) - radius * radius;
                    const float d = b * b - 4.0f * a * c;

                    if (d <= 0) {
                        continue;
                    }
                    const float root = sqrt(d);
                    float t = (-b - root) / (2.0f * a);
                    if (t <= 0.001f) {
                        t = (-b + root) / (2.0f * a);
                        if (t <= 0.001f) {
                            continue;
                        }
                    }

                    if (min_distance <= t) {
                        continue;
                    }
                    min_distance = t;
                    hit_point_x = ray_origin_x + t * ray_direction_x;
                    hit_point_y = ray_origin_y + t * ray_direction_y;
                    hit_point_z = ray_origin_z + t * ray_direction_z;

                    float tmp_x = hit_point_x - center_x;
                    float tmp_y = hit_point_y - center_y;
                    float tmp_z = hit_point_z - center_z;
                    float norm = sqrtf(tmp_x * tmp_x + tmp_y * tmp_y + tmp_z * tmp_z) + 1e-12;

                    hit_face_normal_x = tmp_x / norm;
                    hit_face_normal_y = tmp_y / norm;
                    hit_face_normal_z = tmp_z / norm;

                    material_type = shared_material_types[face_index];

                    hit_color_r = shared_face_colors[face_index * colors_stride + 0];
                    hit_color_g = shared_face_colors[face_index * colors_stride + 1];
                    hit_color_b = shared_face_colors[face_index * colors_stride + 2];

                    did_hit_object = true;
                    continue;
                }
                // http://www.cs.utah.edu/~awilliam/box/box.pdf
                if (object_type == 333) {
                    float _min_x = shared_face_vertices[index + 0];
                    float _min_y = shared_face_vertices[index + 1];
                    float _min_z = shared_face_vertices[index + 2];
                    float _max_x = shared_face_vertices[index + 4];
                    float _max_y = shared_face_vertices[index + 5];
                    float _max_z = shared_face_vertices[index + 6];

                    float min_x = shared_camera_inv_matrix[0][0] * _min_x + shared_camera_inv_matrix[1][0] * _min_y + shared_camera_inv_matrix[2][0] * _min_z + shared_camera_inv_matrix[3][0];
                    float min_y = shared_camera_inv_matrix[0][1] * _min_x + shared_camera_inv_matrix[1][1] * _min_y + shared_camera_inv_matrix[2][1] * _min_z + shared_camera_inv_matrix[3][1];
                    float min_z = shared_camera_inv_matrix[0][2] * _min_x + shared_camera_inv_matrix[1][2] * _min_y + shared_camera_inv_matrix[2][2] * _min_z + shared_camera_inv_matrix[3][2];

                    float max_x = shared_camera_inv_matrix[0][0] * _max_x + shared_camera_inv_matrix[1][0] * _max_y + shared_camera_inv_matrix[2][0] * _max_z + shared_camera_inv_matrix[3][0];
                    float max_y = shared_camera_inv_matrix[0][1] * _max_x + shared_camera_inv_matrix[1][1] * _max_y + shared_camera_inv_matrix[2][1] * _max_z + shared_camera_inv_matrix[3][1];
                    float max_z = shared_camera_inv_matrix[0][2] * _max_x + shared_camera_inv_matrix[1][2] * _max_y + shared_camera_inv_matrix[2][2] * _max_z + shared_camera_inv_matrix[3][2];

                    const bool sign_x = ray_direction_inv_x < 0;
                    const bool sign_y = ray_direction_inv_y < 0;
                    const bool sign_z = ray_direction_inv_z < 0;
                    float tmin, tmax, tymin, tymax, tzmin, tzmax;
                    tmin = ((sign_x ? max_x : min_x) - ray_origin_x) * ray_direction_inv_x;
                    tmax = ((sign_x ? min_x : max_x) - ray_origin_x) * ray_direction_inv_x;
                    tymin = ((sign_y ? max_y : min_y) - ray_origin_y) * ray_direction_inv_y;
                    tymax = ((sign_y ? min_y : max_y) - ray_origin_y) * ray_direction_inv_y;
                    if ((tmin > tymax) || (tymin > tmax)) {
                        continue;
                    }
                    if (tymin > tmin) {
                        tmin = tymin;
                    }
                    if (tymax < tmax) {
                        tmax = tymax;
                    }
                    tzmin = ((sign_z ? max_z : min_z) - ray_origin_z) * ray_direction_inv_z;
                    tzmax = ((sign_z ? min_z : max_z) - ray_origin_z) * ray_direction_inv_z;
                    if ((tmin > tzmax) || (tzmin > tmax)) {
                        continue;
                    }
                    if (tzmin > tmin) {
                        tmin = tzmin;
                    }
                    if (tzmax < tmax) {
                        tmax = tzmax;
                    }
                    material_type = RTX_MATERIAL_TYPE_EMISSIVE;

                    hit_color_r = 1.0f;
                    hit_color_g = 1.0f;
                    hit_color_b = 1.0f;

                    did_hit_object = true;
                    continue;
                }
            }

            if (did_hit_object) {
                ray_origin_x = hit_point_x;
                ray_origin_y = hit_point_y;
                ray_origin_z = hit_point_z;

                if (material_type == RTX_MATERIAL_TYPE_EMISSIVE) {
                    color_r = reflection_decay_r * hit_color_r;
                    color_g = reflection_decay_g * hit_color_g;
                    color_b = reflection_decay_b * hit_color_b;
                    did_hit_light = true;
                    break;
                }

                // detect backface
                // float dot = hit_face_normal_x * ray_direction_x + hit_face_normal_y * ray_direction_y + hit_face_normal_z * ray_direction_z;
                // if (dot > 0.0f) {
                //     hit_face_normal_x *= -1.0f;
                //     hit_face_normal_y *= -1.0f;
                //     hit_face_normal_z *= -1.0f;
                // }

                // diffuse reflection
                float diffuese_x = hiprand_normal(&state);
                float diffuese_y = hiprand_normal(&state);
                float diffuese_z = hiprand_normal(&state);
                const float norm = sqrt(diffuese_x * diffuese_x + diffuese_y * diffuese_y + diffuese_z * diffuese_z);
                diffuese_x /= norm;
                diffuese_y /= norm;
                diffuese_z /= norm;

                float dot = hit_face_normal_x * diffuese_x + hit_face_normal_y * diffuese_y + hit_face_normal_z * diffuese_z;
                if (dot < 0.0f) {
                    diffuese_x = -diffuese_x;
                    diffuese_y = -diffuese_y;
                    diffuese_z = -diffuese_z;
                }
                ray_direction_x = diffuese_x;
                ray_direction_y = diffuese_y;
                ray_direction_z = diffuese_z;

                ray_direction_inv_x = 1.0f / ray_direction_x;
                ray_direction_inv_y = 1.0f / ray_direction_y;
                ray_direction_inv_z = 1.0f / ray_direction_z;

                reflection_decay_r *= hit_color_r;
                reflection_decay_g *= hit_color_g;
                reflection_decay_b *= hit_color_b;
            }
        }

        if (did_hit_light == false) {
            color_r = 0.0f;
            color_g = 0.0f;
            color_b = 0.0f;
        }
        color_per_ray[ray_index * 3 + 0] = color_r;
        color_per_ray[ray_index * 3 + 1] = color_g;
        color_per_ray[ray_index * 3 + 2] = color_b;
    }
}

void rtx_cuda_alloc(
    float*& gpu_rays,
    float*& gpu_face_vertices,
    float*& gpu_face_colors,
    int*& gpu_object_types,
    int*& gpu_material_types,
    float*& gpu_color_per_ray,
    float*& gpu_camera_inv_matrix,
    const float* rays,
    const float* face_vertices,
    const float* face_colors,
    const int* object_types,
    const int* material_types,
    const float* camera_inv_matrix,
    const int num_rays,
    const int rays_stride,
    const int num_faces,
    const int faces_stride,
    const int colors_stride,
    const int num_pixels,
    const int num_rays_per_pixel)
{
    hipMalloc((void**)&gpu_rays, sizeof(float) * num_rays * rays_stride);
    hipMemcpy(gpu_rays, rays, sizeof(float) * num_rays * rays_stride, hipMemcpyHostToDevice);

    hipMalloc((void**)&gpu_face_vertices, sizeof(float) * num_faces * faces_stride);
    hipMemcpy(gpu_face_vertices, face_vertices, sizeof(float) * num_faces * faces_stride, hipMemcpyHostToDevice);

    hipMalloc((void**)&gpu_face_colors, sizeof(float) * num_faces * colors_stride);
    hipMemcpy(gpu_face_colors, face_colors, sizeof(float) * num_faces * colors_stride, hipMemcpyHostToDevice);

    hipMalloc((void**)&gpu_object_types, sizeof(int) * num_faces);
    hipMemcpy(gpu_object_types, object_types, sizeof(int) * num_faces, hipMemcpyHostToDevice);

    hipMalloc((void**)&gpu_material_types, sizeof(int) * num_faces);
    hipMemcpy(gpu_material_types, material_types, sizeof(int) * num_faces, hipMemcpyHostToDevice);

    hipMalloc((void**)&gpu_camera_inv_matrix, sizeof(float) * 16);
    hipMemcpy(gpu_camera_inv_matrix, camera_inv_matrix, sizeof(float) * 16, hipMemcpyHostToDevice);

    hipMalloc((void**)&gpu_color_per_ray, sizeof(float) * num_pixels * 3 * num_rays_per_pixel);
}

void rtx_cuda_copy(
    float*& gpu_rays,
    float*& gpu_face_vertices,
    float*& gpu_camera_inv_matrix,
    const float* rays,
    const float* face_vertices,
    const float* camera_inv_matrix,
    const int num_rays,
    const int rays_stride,
    const int num_faces,
    const int faces_stride)
{
    hipMemcpy(gpu_rays, rays, sizeof(float) * num_rays * rays_stride, hipMemcpyHostToDevice);
    hipMemcpy(gpu_face_vertices, face_vertices, sizeof(float) * num_faces * faces_stride, hipMemcpyHostToDevice);
    hipMemcpy(gpu_camera_inv_matrix, camera_inv_matrix, sizeof(float) * 16, hipMemcpyHostToDevice);
}

void rtx_cuda_delete(
    float*& gpu_rays,
    float*& gpu_face_vertices,
    float*& gpu_face_colors,
    int*& gpu_object_types,
    int*& gpu_material_types,
    float*& gpu_color_per_ray,
    float*& gpu_camera_inv_matrix)
{
    hipFree(gpu_rays);
    hipFree(gpu_face_vertices);
    hipFree(gpu_face_colors);
    hipFree(gpu_object_types);
    hipFree(gpu_material_types);
    hipFree(gpu_color_per_ray);
    hipFree(gpu_camera_inv_matrix);
}

void cuda_device_reset()
{
    hipDeviceReset();
}

void rtx_cuda_ray_tracing_render(
    float*& gpu_rays,
    float*& gpu_face_vertices,
    float*& gpu_face_colors,
    int*& gpu_object_types,
    int*& gpu_material_types,
    float*& gpu_color_per_ray,
    float*& color_per_ray,
    float*& gpu_camera_inv_matrix,
    const int num_rays,
    const int num_faces,
    const int faces_stride,
    const int colors_stride,
    const int path_depth,
    const int num_pixels,
    const int num_rays_per_pixel)
{
    assert(num_rays == num_pixels * num_rays_per_pixel);

    int num_threads = 128;
    int num_blocks = (num_rays - 1) / num_threads + 1;

    num_blocks = 512;

    int num_kernels = 1;
    assert(num_rays % num_kernels == 0);

    int num_rays_per_thread = num_rays / (num_threads * num_blocks * num_kernels) + 1;
    int num_rays_per_kernel = num_rays / num_kernels;

    // printf("rays: %d, rays_per_kernel: %d, num_rays_per_thread: %d\n", num_rays, num_rays_per_kernel, num_rays_per_thread);
    // printf("<<<%d, %d>>>\n", num_blocks, num_threads);

    int thread_offset = 0;
    for (int k = 0; k < num_kernels; k++) {
        render<<<num_blocks, num_threads>>>(
            gpu_rays,
            gpu_face_vertices,
            gpu_face_colors,
            gpu_object_types,
            gpu_material_types,
            gpu_color_per_ray,
            gpu_camera_inv_matrix,
            num_rays_per_thread,
            thread_offset,
            num_rays,
            num_faces,
            faces_stride,
            colors_stride,
            path_depth);
        thread_offset += num_rays_per_kernel;
    }
    hipDeviceSynchronize();

    // hipDeviceProp_t dev;
    // hipGetDeviceProperties(&dev, 0);

    // printf(" device name : %s\n", dev.name);
    // printf(" total global memory : %d (MB)\n", dev.totalGlobalMem/1024/1024);
    // printf(" shared memory / block : %d (KB)\n", dev.sharedMemPerBlock/1024);
    // printf(" register / block : %d\n", dev.regsPerBlock);

    hipError_t status = hipGetLastError();
    if (status != 0) {
        fprintf(stderr, "%s\n", hipGetErrorString(status));
    }
    hipMemcpy(color_per_ray, gpu_color_per_ray, sizeof(float) * num_pixels * 3 * num_rays_per_pixel, hipMemcpyDeviceToHost);
}

// void rtx_launch_test_kernel()
// {
//     clock_t c1, c2;

//     int num_vertices = 100000;
//     float* vertices = (float*)malloc(sizeof(float) * num_vertices * 4);
//     for (int n = 0; n < num_vertices; n++) {
//         vertices[n * 4 + 0] = 0.1f;
//         vertices[n * 4 + 1] = 0.1f;
//         vertices[n * 4 + 2] = 0.1f;
//         vertices[n * 4 + 3] = 0.1f;
//     }
//     float* gpu_vertices;
//     hipMalloc((void**)&gpu_vertices, sizeof(float) * num_vertices * 4);
//     hipMemcpy(gpu_vertices, vertices, sizeof(float) * num_vertices * 4, hipMemcpyHostToDevice);

//     double mean = 0.0f;
//     for (int j = 0; j < 10; j++) {
//         c1 = clock();
//         for (int i = 0; i < 1000; i++) {
//             test_kernel<<<512, 128>>>(gpu_vertices, num_vertices);
//             hipDeviceSynchronize();
//         }
//         c2 = clock();
//         mean += (double)(c2 - c1) / CLOCKS_PER_SEC;
//     }
//     printf("time = %lf[s]\n", mean / 10.0);
//     free(vertices);
//     hipFree(gpu_vertices);

//     size_t pitch;
//     hipMallocPitch((void**)&gpu_vertices, &pitch, sizeof(float) * num_vertices * 4, 1);

//     hipError_t status = hipGetLastError();
//     if (status != 0) {
//         fprintf(stderr, "%s\n", hipGetErrorString(status));
//     }

//     printf("pitch = %d\n", (int)(pitch));
//     hipMemcpy(gpu_vertices, vertices, sizeof(float) * num_vertices * 4, hipMemcpyHostToDevice);

//     status = hipGetLastError();
//     if (status != 0) {
//         fprintf(stderr, "%s\n", hipGetErrorString(status));
//     }

//     mean = 0.0f;
//     for (int j = 0; j < 10; j++) {
//         c1 = clock();
//         for (int i = 0; i < 1000; i++) {
//             test_kernel<<<512, 128>>>(gpu_vertices, num_vertices);
//             hipDeviceSynchronize();
//         }
//         c2 = clock();
//         mean += (double)(c2 - c1) / CLOCKS_PER_SEC;
//     }
//     printf("time = %lf[s]\n", mean / 10.0);
//     free(vertices);
//     hipFree(gpu_vertices);
// }